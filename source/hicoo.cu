#include "hicoo.hpp"

void HicooTensor::freeAllArrays() {
    free(points_h);
    free(blocks_h);
    cudaErrorCheck(hipFree(points_d));
    cudaErrorCheck(hipFree(blocks_d));
}

void HicooTensor::uploadToDevice() {
    cudaErrorCheck(hipFree(points_d));
    cudaErrorCheck(hipMalloc((void **) &d_weight, sizeof(float)));
    cudaErrorCheck(hipMemcpy(points_d, points_h, sizeof(HicooPoint) * num_elements, hipMemcpyHostToDevice));
}

void HicooTensor::downloadToHost() {
    free(points_h);
    points_h = malloc(sizeof(HicooPoint) * num_elements);
    cudaErrorCheck(hipMemcpy(points_h, points_d, sizeof(HicooPoint) * num_elements, hipMemcpyDeviceToHost));
}

DenseMatrixManager HicooTensor::mttkrp(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO

    return ret;
}
