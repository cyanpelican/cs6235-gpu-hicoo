#include "hip/hip_runtime.h"
#include "hicoo.hpp"
#include "coo.hpp"


void HicooTensor::freeAllArrays() {
    DEBUG_PRINT("HT: free all arrays\n");
    freeHostArrays();
    freeDeviceArrays();
}
void HicooTensor::freeHostArrays() {
    DEBUG_PRINT("HT: free host arrays\n");
    DEBUG_PRINT("    - points_h = %p\n", points_h);
    DEBUG_PRINT("    - blocks_h = %p\n", blocks_h);
    free(points_h);
    free(blocks_h);
    points_h = nullptr;
    blocks_h = nullptr;
}
void HicooTensor::freeDeviceArrays() {
    DEBUG_PRINT("HT: free device arrays\n");
    DEBUG_PRINT("    - points_d = %p\n", points_d);
    DEBUG_PRINT("    - blocks_d = %p\n", blocks_d);
    if(points_d != nullptr) // Because the docs lie: "If devPtr is 0, no operation is performed."
        cudaErrorCheck(hipFree(points_d));
    if(blocks_d != nullptr) // Because the docs lie: "If devPtr is 0, no operation is performed."
        cudaErrorCheck(hipFree(blocks_d));
    points_d = nullptr;
    blocks_d = nullptr;
}

void HicooTensor::uploadToDevice() {
    DEBUG_PRINT("HT: upload to device\n");
    assert(points_h != nullptr);
    assert(blocks_h != nullptr);
    freeDeviceArrays();

    cudaErrorCheck(hipMalloc((void **) &points_d, sizeof(HicooPoint) * numPoints));
    assert(points_d != nullptr);
    cudaErrorCheck(hipMemcpy(points_d, points_h, sizeof(HicooPoint) * numPoints, hipMemcpyHostToDevice));

    cudaErrorCheck(hipMalloc((void **) &blocks_d, sizeof(HicooBlock) * (numBlocks+1)));
    assert(blocks_d != nullptr);
    cudaErrorCheck(hipMemcpy(blocks_d, blocks_h, sizeof(HicooBlock) * (numBlocks+1), hipMemcpyHostToDevice));
}

void HicooTensor::downloadToHost() {
    DEBUG_PRINT("HT: download to host\n");
    assert(points_d != nullptr);
    assert(blocks_d != nullptr);
    freeHostArrays();

    points_h = (HicooPoint*)malloc(sizeof(HicooPoint) * numPoints);
    assert(points_h != nullptr);
    cudaErrorCheck(hipMemcpy(points_h, points_d, sizeof(HicooPoint) * numPoints, hipMemcpyDeviceToHost));

    blocks_h = (HicooBlock*)malloc(sizeof(HicooBlock) * (numBlocks+1));
    assert(blocks_h != nullptr);
    cudaErrorCheck(hipMemcpy(blocks_h, blocks_d, sizeof(HicooBlock) * (numBlocks+1), hipMemcpyDeviceToHost));
}


CooTensorManager HicooTensor::toCoo() {
    DEBUG_PRINT("HT: to coo\n");
    CooTensorManager ret;
    assert(0); // TODO
    return ret;
}


DenseMatrixManager HicooTensor::mttkrp_naive_cpu(DenseMatrixManager D, DenseMatrixManager C) {
    /*
     * for each block (except the last)
     *      for each element starting at block address and ending at next block address
     *          l = blockX * blockWidth + pointX
     *          k = blockY * blockHeight + pointY
     *          i = blockZ * blockDepth + pointZ
     *
     *          for j = 1..j
     *              A(i,j) += point.val * C(k,j) + D(l,j)
     * return A
     */

    DenseMatrixManager ret;
    DenseMatrix& a = ret;
    DenseMatrix& c = C;
    DenseMatrix& d = D;

    assert(points_h != nullptr);
    assert(blocks_h != nullptr);
    assert(c.values_h != nullptr);
    assert(d.values_h != nullptr);

    //Naive: each thread is a non-zero
    //optimization: each thread does a few R's

    //Naive implementation:

    DEBUG_PRINT("HICOO: mttkrp naive cpu\n");

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    int I = this->depth, J = d.width, K = this->height, L = this->width;
    DEBUG_PRINT("    - I = %d, J = %d, K = %d, L = %d\n", I, J, K, L);
    assert(d.height == L);
    assert(c.height == K);
    assert(c.width  == J);


    a.setSize(I, J);

    //for each non-zero
    DEBUG_PRINT("    - performing operation\n");

    for (int b = 0; b < this->numBlocks; b++) {
        HicooBlock block = this->access_block(b);
        unsigned long long startBlockAddress = block.blockAddress;
        unsigned long long endBlockAddress = this->access_block(b+1).blockAddress;
        for (unsigned long long index = startBlockAddress; index < endBlockAddress; index++) {
            HicooPoint point = access_point(index);

            int l = block.blockX * this->blockWidth + point.x;
            int k = block.blockY * this->blockHeight + point.y;
            int i = block.blockZ * this->blockDepth + point.z;

            for (int j = 0; j < J; j++) {
                a.access(i,j) += point.value * d.access(l,j) * c.access(k,j);
            }
        }
    }

    return ret;
}
__global__ void mttkrp_naive_gpu_kernel(HicooTensor hicooTensor, DenseMatrix d, DenseMatrix c, DenseMatrix ret);

//wrapper function for the sake of convenience
DenseMatrixManager HicooTensor::mttkrp_naive_gpu(DenseMatrixManager D, DenseMatrixManager C) {
    DEBUG_PRINT("HT: naive mttkrp gpu\n");
    this->uploadToDevice();

    DenseMatrixManager ret;
    DenseMatrix& a = ret;
    DenseMatrix& c = C;
    DenseMatrix& d = D;

    assert(points_d != nullptr);
    assert(blocks_d != nullptr);
    assert(c.values_d != nullptr);
    assert(d.values_d != nullptr);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    int I = this->depth, J = d.width, K = this->height, L = this->width;
    DEBUG_PRINT("    - I = %d, J = %d, K = %d, L = %d\n", I, J, K, L);
    assert(d.height == L);
    assert(c.height == K);
    assert(c.width  == J);


    DEBUG_PRINT("    - setSize_d\n");
    a.setSize_d(I, J);

    //todo: split up the blocks & blocks per threads appropriately
    mttkrp_naive_gpu_kernel<<<ceil(this->numBlocks/64.0), 64>>>(*this, d, c, ret);
    hipDeviceSynchronize();

    ret.tensor->tensor.downloadToHost();
    DEBUG_PRINT("    - done\n");

    return ret;
}

//Not declared as part of the class... Cuda doesn't like it's kernels as part of OOP
__global__ void mttkrp_naive_gpu_kernel(HicooTensor hicooTensor, DenseMatrix d, DenseMatrix c, DenseMatrix ret) {
    /*
     * for each block (except the last)
     *      for each element starting at block address and ending at next block address
     *          l = blockX * blockWidth + pointX
     *          k = blockY * blockHeight + pointY
     *          i = blockZ * blockDepth + pointZ
     *
     *          for j = 1..j
     *              A(i,j) += point.val * C(k,j) + D(l,j)
     * return A
     */

    DenseMatrix& a = ret;

    //Naive: each thread is a block
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < hicooTensor.numBlocks) {
        // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
        int J = d.width;// K = hicooTensor.height, L = hicooTensor.width, I = hicooTensor.depth


        //each thread gets a block
        HicooBlock block = hicooTensor.access_block(index);
        unsigned long long startBlockAddress = block.blockAddress;
        unsigned long long endBlockAddress = hicooTensor.access_block(index + 1).blockAddress;
        for (unsigned long long index = startBlockAddress; index < endBlockAddress; index++) {
            HicooPoint point = hicooTensor.access_point(index);

            int l = block.blockX * hicooTensor.blockWidth + point.x;
            int k = block.blockY * hicooTensor.blockHeight + point.y;
            int i = block.blockZ * hicooTensor.blockDepth + point.z;

            for (int j = 0; j < J; j++) {
                float val = point.value * d.access(l, j) * c.access(k, j);
                atomicAdd(&a.access(i,j), val);
            }
        }
    }
}

DenseMatrixManager HicooTensor::mttkrp_guy1(DenseMatrixManager D, DenseMatrixManager C) {
    DenseMatrixManager ret;
    DenseMatrix& c = C;
    DenseMatrix& d = D;

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager HicooTensor::mttkrp_james1(DenseMatrixManager D, DenseMatrixManager C) {
    DenseMatrixManager ret;
    DenseMatrix& c = C;
    DenseMatrix& d = D;

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}


__global__ void hicoo_kevin1_kernel(DenseMatrix a, HicooTensor b, DenseMatrix d, DenseMatrix c) {
    HicooBlock& ba = b.access_block(blockIdx.x);
    HicooBlock& bb = b.access_block(blockIdx.x+1);

    unsigned int bx = ba.blockX * b.blockWidth;
    unsigned int by = ba.blockY * b.blockHeight;
    unsigned int bz = ba.blockZ * b.blockDepth;

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    for(int e = ba.blockAddress; e < bb.blockAddress; e++) {
        HicooPoint& p = b.access_point(e);
        for(int j = threadIdx.x; j < a.width; j+=32) {
            float val = p.value * d.access(p.x+bx,j) * c.access(p.y+by,j);
            atomicAdd(&a.access(p.z+bz, j), val);
        }
    }
}

DenseMatrixManager HicooTensor::mttkrp_kevin1(DenseMatrixManager D, DenseMatrixManager C) {
    // Has each thread block mapped to a hicoo block (parallelizing blocks across J)
    DEBUG_PRINT("HT: mttkrp kevin1\n");
    DEBUG_PRINT("    - asserts, initialization\n");
    DenseMatrixManager ret;
    DenseMatrix& a = ret;
    DenseMatrix& c = C;
    DenseMatrix& d = D;

    assert(points_d != nullptr);
    assert(blocks_d != nullptr);
    assert(c.values_d != nullptr);
    assert(d.values_d != nullptr);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    int I = this->depth, J = d.width, K = this->height, L = this->width;
    DEBUG_PRINT("    - I = %d, J = %d, K = %d, L = %d\n", I, J, K, L);
    assert(d.height == L);
    assert(c.height == K);
    assert(c.width  == J);


    DEBUG_PRINT("    - malloc output matrix\n");
    a.setSize_d(I, J);

    DEBUG_PRINT("    - do compute on gpu\n");
    hicoo_kevin1_kernel<<<numBlocks, 32>>>(a, *this, d, c);

    DEBUG_PRINT("    - downloading to host\n");
    a.downloadToHost();

    DEBUG_PRINT("    - done\n");
    return ret;
}


__global__ void hicoo_kevin2_kernel(DenseMatrix a, HicooTensor b, DenseMatrix d, DenseMatrix c, int* lut) {
    int bi = lut[blockIdx.x];
    HicooBlock& ba = b.access_block(bi);
    while(ba.blockZ == blockIdx.x) {
        HicooBlock& bb = b.access_block(bi+1);

        unsigned int bx = ba.blockX * b.blockWidth;
        unsigned int by = ba.blockY * b.blockHeight;
        unsigned int bz = ba.blockZ * b.blockDepth;

        // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
        for(int e = ba.blockAddress; e < bb.blockAddress; e++) {
            HicooPoint& p = b.access_point(e);
            for(int j = threadIdx.x; j < a.width; j+=32) {
                float val = p.value * d.access(p.x+bx,j) * c.access(p.y+by,j);
                a.access(p.z+bz, j) += val;
            }
        }

        ba = bb;
        bi++;
    }
}

__global__ void hicoo_kevin2_lut_populate(HicooTensor b, int* lut) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < b.numBlocks && idx > 0) {
        HicooBlock prev = b.access_block(idx-1);
        HicooBlock curr = b.access_block(idx);

        if(prev.blockZ != curr.blockZ) {
            lut[curr.blockZ] = idx;
        }
    }
}

DenseMatrixManager HicooTensor::mttkrp_kevin2(DenseMatrixManager D, DenseMatrixManager C) {
    // Has each thread block mapped to a hicoo block (parallelizing blocks across J)
    DEBUG_PRINT("HT: mttkrp kevin2\n");
    DEBUG_PRINT("    - asserts, initialization\n");
    DenseMatrixManager ret;
    DenseMatrix& a = ret;
    DenseMatrix& c = C;
    DenseMatrix& d = D;

    assert(points_d != nullptr);
    assert(blocks_d != nullptr);
    assert(c.values_d != nullptr);
    assert(d.values_d != nullptr);

    assert(sorting == ZYX);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    int I = this->depth, J = d.width, K = this->height, L = this->width;
    DEBUG_PRINT("    - I = %d, J = %d, K = %d, L = %d\n", I, J, K, L);
    assert(d.height == L);
    assert(c.height == K);
    assert(c.width  == J);

    DEBUG_PRINT("    - malloc output matrix\n");
    a.setSize_d(I, J);

    DEBUG_PRINT("    - create LUT on gpu\n");
    int blocksZ = (width-1)/blockDepth + 1;
    int* zBlockIndices;
    cudaErrorCheck(hipMalloc((void **) &zBlockIndices, sizeof(int) * blocksZ));
    assert(zBlockIndices != nullptr);
    cudaErrorCheck(hipMemset(zBlockIndices, 0, blocksZ * sizeof(int)));

    DEBUG_PRINT("    - populate LUT on gpu\n");
    hicoo_kevin2_lut_populate<<<(numBlocks-1)/32+1, 32>>>(*this, zBlockIndices);

    DEBUG_PRINT("    - do compute on gpu\n");
    hicoo_kevin2_kernel<<<blocksZ, 32>>>(a, *this, d, c, zBlockIndices);

    DEBUG_PRINT("    - Freeing LUT\n");
    cudaErrorCheck(hipFree(zBlockIndices));

    DEBUG_PRINT("    - downloading to host\n");
    a.downloadToHost();

    DEBUG_PRINT("    - done\n");
    return ret;
}



__global__ void hicoo_kevin3_kernel(DenseMatrix a, HicooTensor b, DenseMatrix d, DenseMatrix c) {
    int bi = blockIdx.x;
    HicooBlock& ba = b.access_block(bi);
    if(bi > 0 && ba.blockZ != b.access_block(bi-1).blockZ) {
        return;
    }

    int blockZ = ba.blockZ;

    while(ba.blockZ == blockZ && bi < b.numBlocks) {
        HicooBlock& bb = b.access_block(bi+1);

        unsigned int bx = ba.blockX * b.blockWidth;
        unsigned int by = ba.blockY * b.blockHeight;
        unsigned int bz = ba.blockZ * b.blockDepth;

        // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
        for(int e = ba.blockAddress; e < bb.blockAddress; e++) {
            HicooPoint& p = b.access_point(e);
            for(int j = threadIdx.x; j < a.width; j+=32) {
                float val = p.value * d.access(p.x+bx,j) * c.access(p.y+by,j);
                a.access(p.z+bz, j) += val;
            }
        }

        ba = bb;
        bi++;
    }
}


DenseMatrixManager HicooTensor::mttkrp_kevin3(DenseMatrixManager D, DenseMatrixManager C) {
    // Has each thread block mapped to a hicoo block (parallelizing blocks across J)
    DEBUG_PRINT("HT: mttkrp kevin3\n");
    DEBUG_PRINT("    - asserts, initialization\n");
    DenseMatrixManager ret;
    DenseMatrix& a = ret;
    DenseMatrix& c = C;
    DenseMatrix& d = D;

    assert(points_d != nullptr);
    assert(blocks_d != nullptr);
    assert(c.values_d != nullptr);
    assert(d.values_d != nullptr);

    assert(sorting == ZYX);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    int I = this->depth, J = d.width, K = this->height, L = this->width;
    DEBUG_PRINT("    - I = %d, J = %d, K = %d, L = %d\n", I, J, K, L);
    assert(d.height == L);
    assert(c.height == K);
    assert(c.width  == J);


    DEBUG_PRINT("    - malloc output matrix\n");
    a.setSize_d(I, J);

    DEBUG_PRINT("    - do compute on gpu\n");
    hicoo_kevin3_kernel<<<numBlocks, 32>>>(a, *this, d, c);

    DEBUG_PRINT("    - downloading to host\n");
    a.downloadToHost();

    DEBUG_PRINT("    - done\n");
    return ret;
}
