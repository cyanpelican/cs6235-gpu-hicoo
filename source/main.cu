#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
#include <sstream>
#include "coo.hpp"
#include "common.hpp"
#include "dense.hpp"
#include "hicoo.hpp"

using namespace std;

const int dimSizeI = 30, dimSizeJ = 30, dimSizeK = 30, dimSizeL = 30;
int RANDOM_SEED = 1234;


template <typename classname, typename funcname>
float validateAndTime(classname inputTensor, DenseMatrixManager expected) {
    hipEvent_t timing_start,timing_stop;

    hipEventCreate(&timing_start);
    hipEventCreate(&timing_stop);

    printf("  Calculating MTTKRP on class %s using %s... ", typeid(classname).name(), typeid(funcname).name());
    hipEventRecord(timing_start,0);
    DenseMatrixManager result = inputTensor.tensor->tensor.funcname(D, C);
    hipEventRecord(timing_stop);
    hipEventSynchronize(timing_stop);
    hipEventElapsedTime(&HicooCPUTime,timing_start, timing_stop);
    compareOutput(expected.tensor->tensor, result.tensor->tensor);
}

void compareOutput(DenseMatrix a, DenseMatrix b) {
    bool success = 1;
    for (int i = 0; i < dimSizeI; i++) {
        for (int j = 0; j < dimSizeJ; j++) {
            float mag = abs(a.access(i, j)) + 1e-4;
            if(abs(a.access(i, j) - b.access(i, j)) > mag * 1e-5) {
                printf("\n    Outputs do not match at index (%d,%d): %f vs %f", i,j, a.access(i,j), b.access(i,j));
                success = 0;
            }
        }
    }
    if (success) { printf("Passed.\n"); }
    else { printf("      FAILED :|\n"); }
}

void validateGroundTruth();
void testDenseToCoo(CooTensorManager Coo, int dimSizeI, int dimSizeK, int dimSizeL);

int main(int argc, char *argv[]) {

    int mode = 0;


    if (argc >= 2) {
        mode = 1;
    }

    printf("Creating TensorManager Objects... ");
    CooTensorManager Coo;
    DenseTensorManager B;
    HicooTensorManager Hicoo;
    printf("Done.\n");

    printf("Creating Timing Variables... ");
    hipEvent_t timing_start,timing_stop;

    hipEventCreate(&timing_start);
    hipEventCreate(&timing_stop);
    printf("Done.\n");

    if (mode == 0) {

        printf("No command line arguments detected... Beginning generic testing sequence...\n\n");
        //exit(0);


        printf("Creating Random Dense Tensor (B) for testing... ");
        B.tensor->tensor.setSize(dimSizeI,dimSizeK,dimSizeL);
        srand(RANDOM_SEED);
        for (int i = 0; i < dimSizeI; i++) {
            for (int k = 0; k < dimSizeK; k++) {
                for (int l = 0; l < dimSizeL; l++) {
                     B.tensor->tensor.access(i,k,l) = rand() / (float) RAND_MAX;
                }
            }
        }
        printf("Done.\n");

        printf("Creating CooTensor... ");
        Coo.tensor->tensor.setSize(dimSizeI*dimSizeK*dimSizeL,dimSizeI,dimSizeK,dimSizeL);
        printf("Done.\n");
    }

    if (mode == 1) {
        //NEED TO CREATE TENSOR FROM FILEIN

        printf("Creating CooTensor from file '%s'... ", argv[1]);
        Coo.create(argv[1]);
        dimSizeI = Coo.tensor->tensor.depth;
        dimSizeK = Coo.tensor->tensor.height;
        dimSizeL = Coo.tensor->tensor.width;
        printf("Done.\n");
    }

    printf("=============================== Begin Test ================================\n\n");



    unsigned long long memUsage;


    printf("  Creating Random Dense Matrices (D,C) for testing... ");
    DenseMatrixManager D,C;
    DenseMatrix& c = C;
    DenseMatrix& d = D;
    d.setSize(dimSizeL,dimSizeJ);
    c.setSize(dimSizeK,dimSizeJ);
    for (int l = 0; l < dimSizeL; l++) {
        for (int j = 0; j < dimSizeJ; j++) {
            d.access(l,j) = rand() / (float) RAND_MAX;
        }
    }
    for (int k = 0; k < dimSizeK; k++) {
        for (int j = 0; j < dimSizeJ; j++) {
            c.access(k,j) = rand() / (float) RAND_MAX;
        }
    }
    printf("Done.\n");



    printf("\n=================== Beginning Kernel Tests on COO Tensor ===================\n\n");

    if (mode == 0) {
        testDenseToCoo(Coo, dimSizeI, dimSizeK, dimSizeL)
    }

    memUsage = Coo.tensor->tensor.getTotalMemory();
    printf("(Memory usage: %llu)\n",memUsage);

    printf("  Calculating MTTKRP (Coo) using implemented CPU kernel function call... ");
    //Time Sequential
    float CooCPUTime;
    hipEventRecord(timing_start,0);
    DenseMatrixManager retCooCPU = Coo.tensor->tensor.mttkrp_naive_cpu(D, C);
    hipEventRecord(timing_stop);
    hipEventSynchronize(timing_stop);
    hipEventElapsedTime(&CooCPUTime,timing_start, timing_stop);
    printf("Done.\n");

    printf("\n  Calculating MTTKRP (Coo) using implemented GPU kernel function call... ");
    //Time Parallel
    float CooGPUTime;
    hipEventRecord(timing_start,0);
    DenseMatrixManager retCooGPU = Coo.tensor->tensor.mttkrp_naive_gpu(D, C); //COO GPU KERNEL
    hipEventRecord(timing_stop);
    hipEventSynchronize(timing_stop);
    hipEventElapsedTime(&CooGPUTime,timing_start, timing_stop);
    printf("Done\n");


    if (mode == 0) {
        //DenseMatixManager Variables

        {
            printf("  Comparing Kevin's Dense implementation to CPU Kernel Call (Dense.naive_cpu vs Coo.naive_cpu)... ");
            DenseMatrixManager retDense = B.tensor->tensor.mttkrp_naive_cpu(D,C);
            compareOutput(retDense.tensor->tensor, retCooCPU.tensor->tensor);
        }
    }

    // TODO - this
    float f = validateAndTime<CooTensorManager, mttkrp_naive_gpu>(Coo, retCooCPU);

    {
        printf("\n  Calculating MTTKRP (Coo) using implemented GPU kernel function call... ");
        DenseMatrixManager retCooGpu = Coo.tensor->tensor.mttkrp_naive_gpu(D,C); //COO GPU KERNEL

        printf("  Comparing GPU Kernel Call to Ground Truth (Coo.naive_gpu vs Ground truth)... ");
        compareOutput(retCooCPU.tensor->tensor, retCooGpu.tensor->tensor);
    }


    printf("\n=================== Beginning Kernel Tests on HiCOO Tensor ===================\n\n");

    printf("  Converting to hicoo\n");
    Hicoo = Coo.tensor->tensor.toHicoo();
    float HicooCPUTime;
    {
        //Time Sequential
        printf("  Calculating MTTKRP (HiCOO) using implemented CPU kernel function call... ");
        hipEventRecord(timing_start,0);
        DenseMatrixManager retHicoo = Hicoo.tensor->tensor.mttkrp_naive_cpu(D, C);
        hipEventRecord(timing_stop);
        hipEventSynchronize(timing_stop);
        hipEventElapsedTime(&HicooCPUTime,timing_start, timing_stop);
        compareOutput(retCooCPU.tensor->tensor, retHicoo.tensor->tensor);
    }

    float HicooGPUTime;
    {
        //Time Parallel
        printf("  Comparing Hicoo implementation to CPU Kernel Call (Ground truth vs HiCoo.naive_cpu)... ");
        hipEventRecord(timing_start,0);
        DenseMatrixManager retHicoo = Hicoo.tensor->tensor.mttkrp_naive_gpu(D, C);
        hipEventRecord(timing_stop);
        hipEventSynchronize(timing_stop);
        hipEventElapsedTime(&HicooGPUTime,timing_start,timing_stop);
        compareOutput(retCooCPU.tensor->tensor, retHicoo.tensor->tensor);
    }

    printf("\n  ==================== Memory Usage ======================= \n");

    if (mode == 0) {
        memUsage = B.tensor->tensor.getTotalMemory();
        printf("  Dense Tensor (%d,%d,%d) --> %llu B\n",dimSizeI,dimSizeK,dimSizeL,memUsage);
    }

    memUsage = Coo.tensor->tensor.getTotalMemory();
    printf("  COO Tensor (%d,%d,%d) --> %llu B\n",dimSizeI,dimSizeK,dimSizeL,memUsage);

    memUsage = Hicoo.tensor->tensor.getTotalMemory();
    printf("  HiCOO Tensor (%d,%d,%d) --> %llu B\n",dimSizeI,dimSizeK,dimSizeL,memUsage);

    printf("  =========================================================\n\n");

    printf("\n  ======================= Timing(s) ======================= \n");

    printf("  COO MTTKRP (%d,%d,%d)\n",dimSizeI,dimSizeK,dimSizeL);
    printf("    CPU -> %f\n", CooCPUTime);
    printf("    GPU -> %f\n", CooGPUTime);
    printf("    Speedup -> %f\n\n", CooCPUTime/CooGPUTime);
    printf("  HiCOO MTTKRP (%d,%d,%d)\n",dimSizeI,dimSizeK,dimSizeL);
    printf("    CPU -> %f\n", HicooCPUTime);
    printf("    GPU -> %f\n", HicooGPUTime);
    printf("    Speedup -> %f\n", HicooCPUTime/HicooGPUTime);

    printf("  =========================================================\n\n");
    printf("That's a wrap\n");
    return 0;
}

void testDenseToCoo(CooTensorManager Coo, int dimSizeI, int dimSizeK, int dimSizeL) {
    printf("  Creating CooTensor from known data for comparison... ");
    srand(RANDOM_SEED);
    for (int i = 0; i < dimSizeI; i++) {
        for (int k = 0; k < dimSizeK; k++) {
            for (int l = 0; l < dimSizeL; l++) {
                int idx = i*dimSizeK*dimSizeL + k*dimSizeL + l;
                CooPoint p;
                p.x = l; p.y = k; p.z = i;
                p.value = rand() / (float) RAND_MAX;
                if(p.value > 1e-4) Coo.tensor->tensor.access(idx) = p;
            }
        }
    }
    printf("Done. ");


    printf("  Testing Dense to Coo conversion function... ");
    CooTensorManager CooComp = B.tensor->tensor.toCoo();
    bool mismatch = 0;
    for (int idx = 0; idx < dimSizeI*dimSizeK*dimSizeL; idx++) {
        CooPoint a, b;
        a = Coo.tensor->tensor.access(idx);
        b = CooComp.tensor->tensor.access(idx);
        if (a.x != b.x || a.y != b.y || a.z != b.z || a.value != b.value) {
            mismatch = 1;
            //printf("    idx: %d  x: %d/%d  y: %d/%d  z: %d/%d  val: %d/%d\n",idx,a.x,b.x,a.y,b.y,a.z,b.z,a.value,b.value);
        }
    }
    if (mismatch) { printf("... Failed.\n"); }
    else { printf("Passed.\n"); }

}

void validateGroundTruth() {

    /*=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-   MATLAB TENSOR / MATRIX VALIDATION CODE =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-*/

    // STRUCTURE OF MATLAB TENSOR:

    // y = i, x = j, z = k;
    //    /    j
    //   ============//
    //   ============//
    // i ============// k
    //   ============//
    //   ============/



    DenseTensorManager matlab;
    matlab.tensor->tensor.setSize(3,3,3);

    matlab.tensor->tensor.access(0,0,0) = 0.8311;
    matlab.tensor->tensor.access(0,0,1) = 0.3952;
    matlab.tensor->tensor.access(0,0,2) = 0.4412;

    matlab.tensor->tensor.access(0,1,0) = 0.5568;
    matlab.tensor->tensor.access(0,1,1) = 0.2911;
    matlab.tensor->tensor.access(0,1,2) = 0.2135;

    matlab.tensor->tensor.access(0,2,0) = 0.2345;
    matlab.tensor->tensor.access(0,2,1) = 0.2098;
    matlab.tensor->tensor.access(0,2,2) = 0.1484;

    matlab.tensor->tensor.access(1,0,0) = 0.2844;
    matlab.tensor->tensor.access(1,0,1) = 0.2804;
    matlab.tensor->tensor.access(1,0,2) = 0.0949;

    matlab.tensor->tensor.access(1,1,0) = 0.3379;
    matlab.tensor->tensor.access(1,1,1) = 0.9659;
    matlab.tensor->tensor.access(1,1,2) = 0.7877;

    matlab.tensor->tensor.access(1,2,0) = 0.4038;
    matlab.tensor->tensor.access(1,2,1) = 0.0240;
    matlab.tensor->tensor.access(1,2,2) = 0.6363;

    matlab.tensor->tensor.access(2,0,0) = 0.3720;
    matlab.tensor->tensor.access(2,0,1) = 0.6422;
    matlab.tensor->tensor.access(2,0,2) = 0.0034;

    matlab.tensor->tensor.access(2,1,0) = 0.9030;
    matlab.tensor->tensor.access(2,1,1) = 0.4056;
    matlab.tensor->tensor.access(2,1,2) = 0.8192;

    matlab.tensor->tensor.access(2,2,0) = 0.3261;
    matlab.tensor->tensor.access(2,2,1) = 0.7646;
    matlab.tensor->tensor.access(2,2,2) = 0.5833;


    DenseMatrixManager mD, mC;
    mD.tensor->tensor.setSize(3,3);
    mC.tensor->tensor.setSize(3,3);

    mD.tensor->tensor.access(0,0) = 0.2061;
    mD.tensor->tensor.access(0,1) = 0.8238;
    mD.tensor->tensor.access(0,2) = 0.0042;
    mD.tensor->tensor.access(1,0) = 0.7055;
    mD.tensor->tensor.access(1,1) = 0.7682;
    mD.tensor->tensor.access(1,2) = 0.4294;
    mD.tensor->tensor.access(2,0) = 0.9975;
    mD.tensor->tensor.access(2,1) = 0.3894;
    mD.tensor->tensor.access(2,2) = 0.3276;

    mC.tensor->tensor.access(0,0) = 0.7853;
    mC.tensor->tensor.access(0,1) = 0.9508;
    mC.tensor->tensor.access(0,2) = 0.3240;
    mC.tensor->tensor.access(1,0) = 0.4353;
    mC.tensor->tensor.access(1,1) = 0.7073;
    mC.tensor->tensor.access(1,2) = 0.7889;
    mC.tensor->tensor.access(2,0) = 0.7104;
    mC.tensor->tensor.access(2,1) = 0.1381;
    mC.tensor->tensor.access(2,2) = 0.2877;

    DenseMatrixManager matlabComp = matlab.tensor->tensor.mttkrp_naive_cpu(mD,mC);
    printf("Output of MTTKRP on Dense Matrix from MATLAB values:\n");

    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            printf("%f ", matlabComp.tensor->tensor.access(i,j));
        }
        printf("\n");
    }

    /* =========   OUTPUT FROM ABOVE CALC    =========

    1.175773 1.701301 0.298766
    1.466742 1.484061 0.644793
    1.824243 1.883446 0.592149

    /* =========   MTTKRP CODE FROM MATLAB:  =========

    n = 1
    KRP = khatrirao(D,C); %<--Khatri-Rao product, omitting U{2}
    M = permute(X.data, [n:size(X,n), 1:n-1]);
    M = reshape(M,size(X,n),[]); %<--Matricized tensor data
    M*KRP

    ans =

    1.1757    1.7013    0.2988
    1.4666    1.4841    0.6449
    1.8243    1.8836    0.5922


    exit(0);

    =================================================*/
}
