#include "hip/hip_runtime.h"
#include "dense.hpp"
#include "coo.hpp"

void DenseTensor::freeAllArrays() {
    DEBUG_PRINT("DT: freeing all arrays\n");
    freeHostArrays();
    freeDeviceArrays();
}
void DenseTensor::freeHostArrays() {
    DEBUG_PRINT("DT: freeing host arrays\n");
    DEBUG_PRINT("    - values_h = %p\n", values_h);
    free(values_h);
    values_h = nullptr;
}
void DenseTensor::freeDeviceArrays() {
    DEBUG_PRINT("DT: freeing device arrays\n");
    DEBUG_PRINT("    - values_d = %p\n", values_d);
    if(values_d != nullptr) // Because the docs lie: "If devPtr is 0, no operation is performed."
        cudaErrorCheck(hipFree(values_d));
    values_d = nullptr;
}

// safely uploads to gpu
void DenseTensor::uploadToDevice() {
    DEBUG_PRINT("DT: upload to device\n");
    freeDeviceArrays();
    cudaErrorCheck(hipMalloc((void **) &values_d, sizeof(float) * width*height*depth));
    assert(values_d != nullptr);
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(float) * width*height*depth, hipMemcpyHostToDevice));
}

// safely downloads from gpu
void DenseTensor::downloadToHost() {
    DEBUG_PRINT("DT: download to host\n");
    freeHostArrays();
    values_h = (float*)malloc(sizeof(float) * width*height*depth);
    assert(values_h != nullptr);
    cudaErrorCheck(hipMemcpy(values_h, values_d, sizeof(float) * width*height*depth, hipMemcpyDeviceToHost));
}


void DenseMatrix::freeAllArrays() {
    DEBUG_PRINT("DM: freeing all arrays\n");
    freeHostArrays();
    freeDeviceArrays();
}
void DenseMatrix::freeHostArrays() {
    DEBUG_PRINT("DM: freeing host arrays\n");
    DEBUG_PRINT("    - values_h = %p\n", values_h);
    free(values_h);
    values_h = nullptr;
}
void DenseMatrix::freeDeviceArrays() {
    DEBUG_PRINT("DM: freeing device arrays\n");
    DEBUG_PRINT("    - values_d = %p\n", values_d);
    if(values_d != nullptr) // Because the docs lie: "If devPtr is 0, no operation is performed."
        cudaErrorCheck(hipFree(values_d));
    values_d = nullptr;
}

// safely uploads to gpu
void DenseMatrix::uploadToDevice() {
    DEBUG_PRINT("DM: upload to device\n");
    freeDeviceArrays();
    cudaErrorCheck(hipMalloc((void **) &values_d, sizeof(float) * width*height));
    assert(values_d != nullptr);
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(float) * width*height, hipMemcpyHostToDevice));
}

// safely downloads from gpu
void DenseMatrix::downloadToHost() {
    DEBUG_PRINT("DM: download to host\n");
    freeHostArrays();
    values_h = (float*)malloc(sizeof(float) * width*height);
    assert(values_h != nullptr);
    cudaErrorCheck(hipMemcpy(values_h, values_d, sizeof(float) * width*height, hipMemcpyDeviceToHost));
}


CooTensorManager DenseTensor::toCoo(float epsilon) {
    DEBUG_PRINT("DT: to coo (epsilon = %f)\n", epsilon);
    CooTensorManager ret;
    CooTensor& tensor = ret;

    DEBUG_PRINT("    - count nnz\n");
    unsigned long long numNonzeros = 0;
    for(int i = 0; i < depth; i++) {
        for(int j = 0; j < height; j++) {
            for(int k = 0; k < width; k++) {
                if(abs(access(k, j, i)) > epsilon) {
                    numNonzeros++;
                }
            }
        }
    }

    DEBUG_PRINT("    - realloc\n");
    tensor.setSize(numNonzeros, depth, height, width);
    tensor.sorting = XYZ;

    // convert
    DEBUG_PRINT("    - final conversion\n");
    unsigned long long ptIdx = 0;
    for(int i = 0; i < depth; i++) {
        for(int j = 0; j < height; j++) {
            for(int k = 0; k < width; k++) {
                if(abs(access(k, j, i)) > epsilon) {
                    tensor.access(ptIdx).value = access(i, j, k);
                    tensor.access(ptIdx).x = k;
                    tensor.access(ptIdx).y = j;
                    tensor.access(ptIdx).z = i;
                    ptIdx++;
                }
            }
        }
    }

    return ret;
}


DenseMatrixManager DenseTensor::mttkrp_naive_cpu(DenseMatrix d, DenseMatrix c) {
    DEBUG_PRINT("DT: mttkrp naive cpu\n");
    DEBUG_PRINT("    - asserts, initialization\n");
    DenseMatrixManager ret;
    DenseMatrix& a = ret;
    assert(values_h != nullptr);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    int I = this->depth, J = d.width, K = this->height, L = this->width;
    DEBUG_PRINT("    - I = %d, J = %d, K = %d, L = %d\n", I, J, K, L);
    assert(d.height == L);
    assert(c.height == K);
    assert(c.width  == J);

    DEBUG_PRINT("    - malloc output\n");
    a.setSize(J, I);

    DEBUG_PRINT("    - compute\n");
    for(int i = 0; i < I; i++) {
        for(int j = 0; j < J; j++) {
            for(int k = 0; k < K; k++) {
              for(int l = 0; l < L; l++) {
                  a.access(j, i) += access(l, k, i) * d.access(j, l) * c.access(j, k);
              }
            }
        }
    }


    return ret;
}

DenseMatrixManager DenseTensor::mttkrp_naive_gpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    assert(values_d != nullptr);

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager DenseTensor::mttkrp_fast(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}


void DenseMatrix::setSize_d(unsigned int width, unsigned int height) {
    DEBUG_PRINT("DM: setSize_d (w %d, h %d)\n", width, height);
    freeDeviceArrays();
    cudaErrorCheck(hipMalloc((void **) &values_d, sizeof(float) * width*height));
    assert(values_d != nullptr);
    cudaErrorCheck(hipMemset(values_d, 0.0f, width*height * sizeof(float)));
    this->width = width;
    this->height = height;
}
