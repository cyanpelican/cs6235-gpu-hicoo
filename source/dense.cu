#include "hip/hip_runtime.h"
#include "dense.hpp"
#include <assert.h>
#include "coo.hpp"

void DenseTensor::freeAllArrays() {
    free(values_h);
    if(values_d != nullptr) // Because the docs lie: "If devPtr is 0, no operation is performed."
        cudaErrorCheck(hipFree(values_d));
}

// safely uploads to gpu
void DenseTensor::uploadToDevice() {
    if(values_d != nullptr) // Because the docs lie: "If devPtr is 0, no operation is performed."
        cudaErrorCheck(hipFree(values_d));
    cudaErrorCheck(hipMalloc((void **) &values_d, sizeof(float) * width*height*depth));
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(float) * width*height*depth, hipMemcpyHostToDevice));
}

// safely downloads from gpu
void DenseTensor::downloadToHost() {
    free(values_h);
    values_h = (float*)malloc(sizeof(float) * width*height*depth);
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(float) * width*height*depth, hipMemcpyDeviceToHost));
}


void DenseMatrix::freeAllArrays() {
    free(values_h);
    if(values_d != nullptr) // Because the docs lie: "If devPtr is 0, no operation is performed."
        cudaErrorCheck(hipFree(values_d));
    values_h = nullptr;
    values_d = nullptr;
}

// safely uploads to gpu
void DenseMatrix::uploadToDevice() {
    if(values_d != nullptr) // Because the docs lie: "If devPtr is 0, no operation is performed."
        cudaErrorCheck(hipFree(values_d));
    cudaErrorCheck(hipMalloc((void **) &values_d, sizeof(float) * width*height));
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(float) * width*height, hipMemcpyHostToDevice));
}

// safely downloads from gpu
void DenseMatrix::downloadToHost() {
    free(values_h);
    values_h = (float*)malloc(sizeof(float) * width*height);
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(float) * width*height, hipMemcpyDeviceToHost));
}


CooTensorManager DenseTensor::toCoo(float epsilon) {
    CooTensorManager ret;
    CooTensor tensor = ret;

    // count NNZs
    unsigned long long numNonzeros = 0;
    for(int i = 0; i < depth; i++) {
        for(int j = 0; j < height; j++) {
            for(int k = 0; k < width; k++) {
                if(abs(access(i, j, k)) < epsilon) {
                    numNonzeros++;
                }
            }
        }
    }

    tensor.setSize(numNonzeros, width, height, depth);
    tensor.sorting = XYZ;

    // convert
    for(int i = 0; i < depth; i++) {
        for(int j = 0; j < height; j++) {
            for(int k = 0; k < width; k++) {
                if(abs(access(i, j, k)) < epsilon) {
                    tensor.access(ptIdx++) = access(i, j, k);
                }
            }
        }
    }

    return ret;
}


DenseMatrixManager DenseTensor::mttkrp_naive_cpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    DenseMatrix a = ret;
    assert(values_h != nullptr);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    int I = this->width, J = d.height, K = this->height, L = this->depth;
    assert(d.width  == L);
    assert(c.width  == K);
    assert(c.height == J);

    a.setSize(I, J);

    for(int i = 0; i < I; i++) {
        for(int j = 0; j < J; j++) {
            for(int k = 0; k < K; k++) {
              for(int l = 0; l < L; l++) {
                  a.access(i, j) += access(i,k,l) * d.access(l,j) * c.access(k,j);
              }
            }
        }
    }


    return ret;
}

DenseMatrixManager DenseTensor::mttkrp_naive_gpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    assert(values_d != nullptr);

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager DenseTensor::mttkrp_fast(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}
