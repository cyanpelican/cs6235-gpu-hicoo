#include "dense.hpp"
#include <assert.h>

void DenseTensor::freeAllArrays() {
    free(values_h);
    cudaErrorCheck(hipFree(values_d));
}

// safely uploads to gpu
void DenseTensor::uploadToDevice() {
    cudaErrorCheck(hipFree(values_d));
    cudaErrorCheck(hipMalloc((void **) &values_d, sizeof(float) * width*height*depth));
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(float) * width*height*depth, hipMemcpyHostToDevice));
}

// safely downloads from gpu
void DenseTensor::downloadToHost() {
    free(values_h);
    values_h = (float*)malloc(sizeof(float) * width*height*depth);
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(float) * width*height*depth, hipMemcpyDeviceToHost));
}


void DenseMatrix::freeAllArrays() {
    free(values_h);
    cudaErrorCheck(hipFree(values_d));
}

// safely uploads to gpu
void DenseMatrix::uploadToDevice() {
    cudaErrorCheck(hipFree(values_d));
    cudaErrorCheck(hipMalloc((void **) &values_d, sizeof(float) * width*height));
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(float) * width*height, hipMemcpyHostToDevice));
}

// safely downloads from gpu
void DenseMatrix::downloadToHost() {
    free(values_h);
    values_h = (float*)malloc(sizeof(float) * width*height);
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(float) * width*height, hipMemcpyDeviceToHost));
}

DenseMatrixManager DenseTensor::mttkrp_naive_cpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    DenseMatrix a = ret;
    assert(values_h != nullptr);

    // TODO - remalloc arrays
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    int I = 0, J = 0, K = 0, L = 0;
    for(int i = 0; i < I; i++) {
        for(int j = 0; j < J; j++) {
            for(int k = 0; k < K; k++) {
              for(int l = 0; l < L; l++) {
                  a.access(i, j) += access(i,j,k) * d.access(l,j) * c.access(k,j);
              }
            }
        }
    }


    return ret;
}

DenseMatrixManager DenseTensor::mttkrp_naive_gpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    assert(values_d != nullptr);

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager DenseTensor::mttkrp_fast(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}
