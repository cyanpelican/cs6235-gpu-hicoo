#include "dense.hpp"

void DenseTensor::freeAllArrays() {
    free(values_h);
    cudaErrorCheck(hipFree(values_d));
}

// safely uploads to gpu
void DenseTensor::uploadToDevice() {
    cudaErrorCheck(hipFree(values_d));
    cudaErrorCheck(hipMalloc((void **) &d_weight, sizeof(float)));
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(DensePoint) * num_elements, hipMemcpyHostToDevice));
}

// safely downloads from gpu
void DenseTensor::downloadToHost() {
    free(values_h);
    values_h = malloc(sizeof(DensePoint) * num_elements);
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(DensePoint) * num_elements, hipMemcpyDeviceToHost));
}


void DenseMatrix::freeAllArrays() {
    free(values_h);
    cudaErrorCheck(hipFree(values_d));
}

// safely uploads to gpu
void DenseMatrix::uploadToDevice() {
    cudaErrorCheck(hipFree(values_d));
    cudaErrorCheck(hipMalloc((void **) &d_weight, sizeof(float)));
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(CooPoint) * num_elements, hipMemcpyHostToDevice));
}

// safely downloads from gpu
void DenseMatrix::downloadToHost() {
    free(values_h);
    values_h = malloc(sizeof(CooPoint) * num_elements);
    cudaErrorCheck(hipMemcpy(values_d, values_h, sizeof(CooPoint) * num_elements, hipMemcpyDeviceToHost));
}

DenseMatrixManager DenseTensor::mttkrp_naive_cpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO

    return ret;
}

DenseMatrixManager DenseTensor::mttkrp_naive_gpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO

    return ret;
}

DenseMatrixManager DenseTensor::mttkrp_fast(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO

    return ret;
}
