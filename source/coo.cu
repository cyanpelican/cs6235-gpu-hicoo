#include "coo.hpp"
#include <assert.h>
#include "csf.hpp"
#include "hicoo.hpp"

void CooTensor::freeAllArrays() {
    free(points_h);
    cudaErrorCheck(hipFree(points_d));
    points_h = nullptr;
    points_d = nullptr;
}

void CooTensor::uploadToDevice() {
    cudaErrorCheck(hipFree(points_d));
    cudaErrorCheck(hipMalloc((void **) &points_d, sizeof(CooPoint) * numElements));
    cudaErrorCheck(hipMemcpy(points_d, points_h, sizeof(CooPoint) * numElements, hipMemcpyHostToDevice));
}

void CooTensor::downloadToHost() {
    free(points_h);
    points_h = (CooPoint*)malloc(sizeof(CooPoint) * numElements);
    cudaErrorCheck(hipMemcpy(points_h, points_d, sizeof(CooPoint) * numElements, hipMemcpyDeviceToHost));
}


HicooTensorManager CooTensor::toHicoo() {
    HicooTensorManager ret;
    assert(0);
    return ret;
}
DenseTensorManager CooTensor::toDense() {
    DenseTensorManager ret;
    assert(0);
    return ret;
}
CsfTensorManager CooTensor::toCsf() {
    CsfTensorManager ret;
    assert(0);
    return ret;
}


DenseMatrixManager CooTensor::mttkrp_naive_cpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    assert(points_h != nullptr);

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager CooTensor::mttkrp_naive_gpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    assert(points_d != nullptr);

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager CooTensor::mttkrp_fast(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}
