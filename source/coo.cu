#include "coo.hpp"

void CooTensor::freeAllArrays() {
    free(points_h);
    cudaErrorCheck(hipFree(points_d));
}

void CooTensor::uploadToDevice() {
    cudaErrorCheck(hipFree(points_d));
    cudaErrorCheck(hipMalloc((void **) &d_weight, sizeof(float)));
    cudaErrorCheck(hipMemcpy(points_d, points_h, sizeof(CooPoint) * num_elements, hipMemcpyHostToDevice));
}

void CooTensor::downloadToHost() {
    free(points_h);
    points_h = malloc(sizeof(CooPoint) * num_elements);
    cudaErrorCheck(hipMemcpy(points_h, points_d, sizeof(CooPoint) * num_elements, hipMemcpyDeviceToHost));
}

DenseMatrixManager CooTensor::mttkrp(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO

    return ret;
}
