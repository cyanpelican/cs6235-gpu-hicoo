#include "hip/hip_runtime.h"
#include <map>
#include <math.h>
#include "coo.hpp"
#include "hicoo.hpp"
#include <map>
#include "common.hpp"
#include <list>


void CooTensor::freeAllArrays() {
    DEBUG_PRINT("CT: freeing all arrays\n");
    freeHostArrays();
    freeDeviceArrays();
}
void CooTensor::freeHostArrays() {
    DEBUG_PRINT("CT: freeing host arrays\n");
    DEBUG_PRINT("    - points_h = %p\n", points_h);
    free(points_h);
    points_h = nullptr;
}
void CooTensor::freeDeviceArrays() {
    DEBUG_PRINT("CT: freeing device arrays\n");
    DEBUG_PRINT("    - points_d = %p\n", points_d);
    if(points_d != nullptr) // Because the docs lie: "If devPtr is 0, no operation is performed."
        cudaErrorCheck(hipFree(points_d));
    points_d = nullptr;
}

void CooTensor::uploadToDevice() {
    DEBUG_PRINT("CT: uploading to device\n");
    assert(points_h != nullptr);
    freeDeviceArrays();
    assert(numElements != 0);
    cudaErrorCheck(hipMalloc((void **) &points_d, sizeof(CooPoint) * numElements));
    assert(points_d != nullptr);
    cudaErrorCheck(hipMemcpy(points_d, points_h, sizeof(CooPoint) * numElements, hipMemcpyHostToDevice));
}

// todo: is this ever necessary? The contents of the tensor are never changed. We should probably just free the device
//  memory
void CooTensor::downloadToHost() {
    DEBUG_PRINT("CT: downloading to host\n");
    assert(points_d != nullptr);
    freeHostArrays();
    points_h = (CooPoint*)malloc(sizeof(CooPoint) * numElements);
    assert(points_h != nullptr);
    cudaErrorCheck(hipMemcpy(points_h, points_d, sizeof(CooPoint) * numElements, hipMemcpyDeviceToHost));
}



// for std::map / std::set insertion
bool operator<(const HicooBlock& a, const HicooBlock& b) {
    if(a.blockX < b.blockX) {
        return true;
    } else if(a.blockX > b.blockX) {
        return false;
    }
    if(a.blockY < b.blockY) {
        return true;
    } else if(a.blockY > b.blockY) {
        return false;
    }
    if(a.blockZ < b.blockZ) {
        return true;
    } else if(a.blockZ > b.blockZ) {
        return false;
    }

    return false;
}
HicooTensorManager CooTensor::toHicoo(int blockDepth, int blockHeight, int blockWidth) {
    DEBUG_PRINT("CT: to hicoo (bd %d, bh %d, bw %d)\n", blockDepth, blockHeight, blockWidth);
    HicooTensorManager ret;
    HicooTensor& retTensor = ret;

    // build an std::map of everything
    DEBUG_PRINT("    - building map\n");
    std::map<HicooBlock, std::list<HicooPoint>> hicooMap;
    for(int i = 0; i < numElements; i++) {
        CooPoint p = access(i);

        HicooBlock block = {/*blockAddress =*/ 0,
            /*blockX =*/ (p.x)/blockWidth, /*blockX =*/ (p.y)/blockHeight, /*blockX =*/ (p.z)/blockDepth,
            /*UNUSED =*/ 0};

        HicooPoint hp = {/*x =*/ (unsigned char)((p.x)%blockWidth), /*y =*/ (unsigned char)((p.y)%blockHeight), /*z =*/ (unsigned char)((p.z)%blockDepth),
            /*UNUSED =*/ 0,
            /*value =*/ p.value};

        hicooMap[block].push_back(hp);
    }

    // put everything into the tensor
    DEBUG_PRINT("    - realloc ret tensor\n");
    retTensor.setSize(hicooMap.size(), numElements, depth, height, width, blockDepth, blockHeight, blockWidth);

    unsigned int blockIndex = 0;
    unsigned long long pointIndex = 0;
    DEBUG_PRINT("    - insert to ret tensor\n");
    for(const std::pair<HicooBlock, std::list<HicooPoint>>& pair : hicooMap) {
        retTensor.blocks_h[blockIndex] = pair.first;
        retTensor.blocks_h[blockIndex].blockAddress = pointIndex;
        for(HicooPoint p : pair.second) {
            retTensor.points_h[pointIndex++] = p;
        }
        blockIndex++;
    }

    // final element off the end of the list
    retTensor.blocks_h[blockIndex].blockAddress = pointIndex;
    retTensor.blocks_h[blockIndex].blockX = 0xFFFFFFFF;
    retTensor.blocks_h[blockIndex].blockY = 0xFFFFFFFF;
    retTensor.blocks_h[blockIndex].blockZ = 0xFFFFFFFF;
    retTensor.blocks_h[blockIndex].UNUSED = 0xFFFFFFFF;


    return ret;
}
DenseTensorManager CooTensor::toDense() {
    DEBUG_PRINT("CT: to dense\n");
    DEBUG_PRINT("    - realloc\n");
    DenseTensorManager ret;
    DenseTensor& retTensor = ret;
    retTensor.setSize(depth, height, width);

    DEBUG_PRINT("    - insertion\n");
    for(int i = 0; i < numElements; i++) {
        CooPoint p = access(i);
        retTensor.access(p.z, p.y, p.x) = p.value;
    }

    return ret;
}


DenseMatrixManager CooTensor::mttkrp_naive_cpu(DenseMatrixManager D, DenseMatrixManager C) {
    /*
      * for each non-zero
      *      i = nnz.i, l = nnz.l, k = nnz.k
      *      for j = 1..j
      *          A(i,j) += val(nnz) * C(k,j) * D (l,j)
      *
      * return A
      */

    //Naive: each thread is a non-zero
    //optimization: each thread does a few R's

    //Naive implementation:

    DEBUG_PRINT("COO: mttkrp naive cpu\n");

    DenseMatrixManager ret;
    DenseMatrix& a = ret;
    DenseMatrix& c = C;
    DenseMatrix& d = D;

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    int I = this->depth, J = d.width, K = this->height, L = this->width;
    DEBUG_PRINT("    - I = %d, J = %d, K = %d, L = %d\n", I, J, K, L);
    assert(d.height == L);
    assert(c.height == K);
    assert(c.width  == J);

    assert(points_h != nullptr);
    assert(c.values_h != nullptr);
    assert(d.values_h != nullptr);


    a.setSize(I, J);

    //for each non-zero
    DEBUG_PRINT("    - performing operation\n");
    for (unsigned int index = 0; index < this->numElements; index++) {
        CooPoint point = this->access(index);
        int l = point.x;
        int k = point.y;
        int i = point.z;

        for (int j = 0; j < J; j++) {
            a.access(i,j) += point.value * d.access(l,j) * c.access(k,j);
        }
    }
    DEBUG_PRINT("    - done\n");

    return ret;
}


//Not declared as part of the class... Cuda doesn't like it's kernels as part of OOP
__global__ void mttkrp_naive_gpu_kernel(CooTensor cooTensor, DenseMatrix d, DenseMatrix c, DenseMatrix ret) {
    /*
     * for each non-zero
     *      i = nnz.i, l = nnz.l, k = nnz.k
     *      for j = 1..j
     *          A(i,j) += val(nnz) * C(k,j) * D (l,j)
     *
     * return A
     */

    //Naive: each thread is a non-zero
    //optimization: each thread does a few R's

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    // int I = cooTensor.depth, J = d.width, K = cooTensor.height, L = cooTensor.width;
    int J = d.width;

    //for each non-zero
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < cooTensor.numElements) {
        CooPoint point = cooTensor.access(index);
        int l = point.x;
        int k = point.y;
        int i = point.z;

        for (int j = 0; j < J; j++) {
            float val = point.value * c.access(k, j) * d.access(l, j);
            atomicAdd(&ret.access(i, j), val);
        }
    }
}


//wrapper function for the sake of convenience
DenseMatrixManager CooTensor::mttkrp_naive_gpu(DenseMatrixManager D, DenseMatrixManager C) {
    DEBUG_PRINT("CT: naive mttkrp gpu\n");
    this->uploadToDevice();

    DenseMatrixManager ret;
    DenseMatrix& a = ret;
    DenseMatrix& c = C;
    DenseMatrix& d = D;

    assert(points_h != nullptr);

    int I = this->depth, J = d.width, K = this->height, L = this->width;
    DEBUG_PRINT("    - I = %d, J = %d, K = %d, L = %d\n", I, J, K, L);
    assert(d.height == L);
    assert(c.height == K);
    assert(c.width  == J);
    a.setSize_d(I, J);
    d.uploadToDevice();
    c.uploadToDevice();

    //todo: split up the blocks & blocks per threads appropriately
    mttkrp_naive_gpu_kernel<<<ceil(this->numElements/64.0), 64>>>(*this, d, c, ret);
    hipDeviceSynchronize();

    ret.tensor->tensor.downloadToHost();

    DEBUG_PRINT("    - done\n");
    return ret;
}


DenseMatrixManager CooTensor::mttkrp_guy1(DenseMatrixManager d, DenseMatrixManager c) {
    DEBUG_PRINT("CT: fast mttkrp gpu\n");
    DenseMatrixManager ret;

    // TODO or DELTEME
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager CooTensor::mttkrp_james1(DenseMatrixManager d, DenseMatrixManager c) {
    DEBUG_PRINT("CT: fast mttkrp gpu\n");
    DenseMatrixManager ret;

    // TODO or DELTEME
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}


__global__ void coo_mttkrp_kevin1_kernel(DenseMatrix a, CooTensor b, DenseMatrix d, DenseMatrix c) {
    CooPoint point = b.access(blockIdx.x);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    for(int j = threadIdx.x; j < a.width; j += 32) {
        float val = point.value * d.access(point.x, j) * c.access(point.y, j);
        atomicAdd(&a.access(point.z, j), val);
    }
}

DenseMatrixManager CooTensor::mttkrp_kevin1(DenseMatrixManager D, DenseMatrixManager C) {
    // Has each thread block mapped to a point (parallelizing blocks across J)
    DEBUG_PRINT("CT: mttkrp kevin1\n");
    DEBUG_PRINT("    - asserts, initialization\n");
    DenseMatrixManager ret;
    DenseMatrix& a = ret;
    DenseMatrix& c = C;
    DenseMatrix& d = D;

    assert(points_h != nullptr);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);
    int I = this->depth, J = d.width, K = this->height, L = this->width;
    DEBUG_PRINT("    - I = %d, J = %d, K = %d, L = %d\n", I, J, K, L);
    assert(d.height == L);
    assert(c.height == K);
    assert(c.width  == J);


    DEBUG_PRINT("    - uploadToDevice\n");
    this->uploadToDevice();
    d.uploadToDevice();
    c.uploadToDevice();

    DEBUG_PRINT("    - malloc output matrix\n");
    a.setSize_d(I, J);

    DEBUG_PRINT("    - do compute on gpu\n");
    coo_mttkrp_kevin1_kernel<<<numElements, 32>>>(a, *this, d, c);

    DEBUG_PRINT("    - downloading\n");
    a.downloadToHost();
    DEBUG_PRINT("    - done\n");



    return ret;
}








// File IO stuff
#include <fcntl.h>
struct SplitLine {
    int nwords;
    int word_indices[16];
    char* line; // do not free

    char* word(int i) {
        if(i == -1) {
            return &line[word_indices[nwords-1]];
        }
        return &line[word_indices[i]];
    }
};

struct FastFilestream {
    // loosely based on https://stackoverflow.com/questions/17925051/fast-textfile-reading-in-c
    const static int BUFFER_SIZE = 1024*1024;
    const static int REFILL_THRESHOLD = 1024;

    char buffer[BUFFER_SIZE];
    FILE *fp;
    int idx = 0;
    int end = 0;
    bool dead = false;

    bool nextLine(SplitLine& line) {
        DEBUG_PRINT("NL\n");
        if(end - idx < REFILL_THRESHOLD && !dead) {
            DEBUG_PRINT("Refill\n");
            // if we want to read and are not at the end, memcpy to beginning
            if(idx != 0) {
                DEBUG_PRINT("Shift\n");
                for(int i = idx; i < end; i++) {
                    buffer[i-idx] = buffer[i];
                }

                end = end-idx;
                idx = 0;
            }

            // try to read
            DEBUG_PRINT("Read\n");
            size_t nread = fread(&buffer[end], sizeof(char), BUFFER_SIZE - end, fp);

            // check errors
            DEBUG_PRINT("Check err\n");
            if(nread == 0) {
                // file out of remaining content
                DEBUG_PRINT("Out of remaining\n");
                dead = true;
            } else if(nread == (size_t)-1) {
                // read failed;
                DEBUG_PRINT("Failed\n");
                dead = true;
            } else {
                DEBUG_PRINT("Increment by %llu\n", nread);
                end += nread;
                DEBUG_PRINT("Now: %d\n", end);
            }

        }

        DEBUG_PRINT("Check completely dead\n");
        // if dead and out of stuff to read
        if(idx >= end) {
            DEBUG_PRINT("Completely dead :(\n");
            return false;
        }
        DEBUG_PRINT("Alive\n");

        // else, do the operation
        line.nwords = 1;
        line.word_indices[0] = 0;
        line.line = &buffer[idx];
        int i;
        for(i = idx; i < end; i++) {
            DEBUG_PRINT("Check char %c\n", buffer[i]);
            if(buffer[i] == ' ') {
                line.word_indices[line.nwords++] = i-idx;
                buffer[i] = 0;
                if(line.nwords >= 15) {
                    break;
                }
            } else if(buffer[i] == '\n') {
                buffer[i] = 0;
                line.word_indices[line.nwords] = i-idx;
                break;
            }
        }
        DEBUG_PRINT("i = %d\n", i);
        idx = i;

        return true;
    }

    FastFilestream(char* fname) {
        //f = open(fname, O_RDONLY);
        fp = fopen(fname, "r");
        assert(fp != nullptr);

        posix_fadvise(fileno(fp), 0, 0, POSIX_FADV_SEQUENTIAL);
    }
    ~FastFilestream() {
        fclose(fp);
    }
};

void CooTensorManager::create(char *tensorFileName) {
    // nell-2 expected: 76879419, d 28819, h 9185, w 12093
    // matmul_2-2-2.tns expected: 8, d 5, h 5, w 5
    DEBUG_PRINT("CT: parsing file %s\n", tensorFileName);
    DEBUG_PRINT("    - file validations, etc\n");
    std::vector<CooPoint> tensorPoints;

    size_t nonZeroes = 0;
    std::ifstream myfile(tensorFileName);
    assert(myfile.good()); // assert file exists, etc

    FastFilestream ffs(tensorFileName);
    SplitLine line;

    //put all the points into a vector
    DEBUG_PRINT("    - load all points into vector\n");
    int maxX = 0, maxY = 0, maxZ = 0;
    while (ffs.nextLine(line)) {
        if(line.nwords < 4 || line.line[0] == '#') {
            // uselessly-short line or comment
            continue;
        }

        printf("        %d words: ", line.nwords);
        for(int i = 0; i < line.nwords; i++) {
            printf("%s ", line.word(i));
        }
        fflush(stdout);

        ++nonZeroes;
        CooPoint currentPoint;
        currentPoint.x = atoi(line.word(0));
        currentPoint.y = atoi(line.word(1));
        currentPoint.z = atoi(line.word(2));
        currentPoint.value = atof(line.word(-1));

        if(currentPoint.x > maxX) maxX = currentPoint.x;
        if(currentPoint.y > maxY) maxY = currentPoint.y;
        if(currentPoint.z > maxZ) maxZ = currentPoint.z;

        //This assumes there are 3 dimensions followed by one value
        tensorPoints.push_back(currentPoint);
    }

    if(tensorPoints.size() != 0)
        DEBUG_PRINT("    - Finished reading; first = (%d,%d,%d)->%f", tensorPoints[0].x, tensorPoints[0].y, tensorPoints[0].z, tensorPoints[0].value);

    //construct the COO object
    DEBUG_PRINT("    - rebuild tensor from input\n");
    tensor->tensor.setSize(nonZeroes, maxZ+1, maxY+1, maxX+1);
    memcpy(tensor->tensor.points_h, tensorPoints.data(), sizeof(CooPoint) * tensorPoints.size());

    DEBUG_PRINT("    - done; size = %d; %d x %d x %d\n", nonZeroes, maxZ, maxY, maxX);
}
