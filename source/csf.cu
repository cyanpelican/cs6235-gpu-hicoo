#include "csf.hpp"
#include <assert.h>

void CsfTensor::freeAllArrays() {
    free(points_h);
    free(fiberAddresses_h);
    cudaErrorCheck(hipFree(points_d));
    cudaErrorCheck(hipFree(fiberAddresses_d));
}

void CsfTensor::uploadToDevice() {
    cudaErrorCheck(hipFree(points_d));
    cudaErrorCheck(hipMalloc((void **) &points_d, sizeof(CsfPoint) * numPoints()));
    cudaErrorCheck(hipMemcpy(points_d, points_h, sizeof(CsfPoint) * numPoints(), hipMemcpyHostToDevice));
    cudaErrorCheck(hipFree(fiberAddresses_d));
    cudaErrorCheck(hipMalloc((void **) &fiberAddresses_d, sizeof(CsfPoint) * (numFibers()+1)));
    cudaErrorCheck(hipMemcpy(fiberAddresses_d, fiberAddresses_h, sizeof(CsfPoint) * (numFibers()+1), hipMemcpyHostToDevice));
}

void CsfTensor::downloadToHost() {
    free(points_h);
    points_h = (CsfPoint*)malloc(sizeof(CsfPoint) * numPoints());
    cudaErrorCheck(hipMemcpy(points_h, points_d, sizeof(CsfPoint) * numPoints(), hipMemcpyDeviceToHost));
    free(fiberAddresses_h);
    fiberAddresses_h = (unsigned int*)malloc(sizeof(unsigned int) * (numFibers()+1));
    cudaErrorCheck(hipMemcpy(fiberAddresses_h, fiberAddresses_d, sizeof(CsfPoint) * (numFibers()+1), hipMemcpyDeviceToHost));
}

DenseMatrixManager CsfTensor::mttkrp_naive_cpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    assert(points_h != nullptr);
    assert(fiberAddresses_h != nullptr);

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager CsfTensor::mttkrp_naive_gpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    assert(points_d != nullptr);
    assert(fiberAddresses_d != nullptr);

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager CsfTensor::mttkrp_fast(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}
