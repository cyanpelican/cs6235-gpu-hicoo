#include "csf.hpp"
#include <assert.h>

void CsfTensor::freeAllArrays() {
    free(points_h);
    free(fiberAddresses_h);
    cudaErrorCheck(hipFree(points_d));
    cudaErrorCheck(hipFree(fiberAddresses_d));
}

void CsfTensor::uploadToDevice() {
    cudaErrorCheck(hipFree(points_d));
    cudaErrorCheck(hipMalloc((void **) &d_weight, sizeof(float)));
    cudaErrorCheck(hipMemcpy(points_d, points_h, sizeof(CsfPoint) * num_elements, hipMemcpyHostToDevice));
}

void CsfTensor::downloadToHost() {
    free(points_h);
    points_h = malloc(sizeof(CsfPoint) * num_elements);
    cudaErrorCheck(hipMemcpy(points_h, points_d, sizeof(CsfPoint) * num_elements, hipMemcpyDeviceToHost));
}

DenseMatrixManager CsfTensor::mttkrp_naive_cpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    assert(points_h != nullptr);
    assert(fiberAddresses_h != nullptr);

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager CsfTensor::mttkrp_naive_gpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;
    assert(points_d != nullptr);
    assert(fiberAddresses_d != nullptr);

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}

DenseMatrixManager CsfTensor::mttkrp_fast(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO
    assert(0);

    // A(i,j) = B(i,k,l) * D(l,j) * C(k,j);

    return ret;
}
