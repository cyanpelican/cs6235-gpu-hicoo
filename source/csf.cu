#include "csf.hpp"

void CsfTensor::freeAllArrays() {
    free(points_h);
    free(fiberAddresses_h);
    cudaErrorCheck(hipFree(points_d));
    cudaErrorCheck(hipFree(fiberAddresses_d));
}

void CsfTensor::uploadToDevice() {
    cudaErrorCheck(hipFree(points_d));
    cudaErrorCheck(hipMalloc((void **) &d_weight, sizeof(float)));
    cudaErrorCheck(hipMemcpy(points_d, points_h, sizeof(CsfPoint) * num_elements, hipMemcpyHostToDevice));
}

void CsfTensor::downloadToHost() {
    free(points_h);
    points_h = malloc(sizeof(CsfPoint) * num_elements);
    cudaErrorCheck(hipMemcpy(points_h, points_d, sizeof(CsfPoint) * num_elements, hipMemcpyDeviceToHost));
}

DenseMatrixManager CsfTensor::mttkrp_naive_cpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO

    return ret;
}

DenseMatrixManager CsfTensor::mttkrp_naive_gpu(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO

    return ret;
}

DenseMatrixManager CsfTensor::mttkrp_fast(DenseMatrix d, DenseMatrix c) {
    DenseMatrixManager ret;

    // TODO

    return ret;
}
